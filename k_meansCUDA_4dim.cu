#include "hip/hip_runtime.h"
#include "k_meansCUDA_4dim.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include ""
#include "k_means_4dim.h"

#define tile 32

__device__ double calculatedistanceGPU(unit* point1, unit* point2) {
	double dis1 = (point1->dim1 - point2->dim1) * (point1->dim1 - point2->dim1);
	double dis2 = (point1->dim2 - point2->dim2) * (point1->dim2 - point2->dim2);
	double dis3 = (point1->dim3 - point2->dim3) * (point1->dim3 - point2->dim3);
	double dis4 = (point1->dim4 - point2->dim4) * (point1->dim4 - point2->dim4);

	return (double)sqrt(dis1 + dis2 + dis3 + dis4);
	//return (double)sqrt((double)pow(point1->dim1 - point2->dim1, 2) + (double)pow(point1->dim2 - point2->dim2, 2) + (double)pow(point1->dim3 - point2->dim3, 2) + (double)pow(point1->dim4 - point2->dim4, 2));
}

__global__ void closestcentroidGPU(unit* points, unit* centroids, int numofcentr, int numofpoints) {

	int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
	int threadPosInBlock = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;
	int blockPosInGrid = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;
	int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;

	if (tid < numofpoints) {
		double dist = 0;
		double firstdistance = calculatedistanceGPU(&points[tid], &centroids[0]);
		points[tid].cluster = 0;
		for (int i = 1; i < numofcentr; i++) {
			dist = calculatedistanceGPU(&points[tid], &centroids[i]);
			if (dist <= firstdistance) {
				points[tid].cluster = i;
				firstdistance = dist;
			}
		}
	}
}

__global__ void closestcentroidSharedGPU(unit* points, unit* centroids, int numofcentr, int numofpoints) {

	int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
	int threadPosInBlock = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;
	int blockPosInGrid = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;
	int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;

	__shared__ unit sh_points[tile*tile];
	__shared__ unit sh_centrs[4];
	
	if (tid < numofpoints) {
		sh_points[threadPosInBlock] = points[tid];
		if (tid%threadsPerBlock ==0) {
			for (int i = 0; i < numofcentr; i++) {
				sh_centrs[i] = centroids[i];
			}
			
		}
		__syncthreads();

		double dist = 0;
		double firstdistance = calculatedistanceGPU(&sh_points[threadPosInBlock], &sh_centrs[0]);
		sh_points[threadPosInBlock].cluster = 0;
		
		for (int i = 1; i < numofcentr; i++) {
			dist = calculatedistanceGPU(&sh_points[threadPosInBlock], &sh_centrs[i]);
			if (dist <= firstdistance) {
				sh_points[threadPosInBlock].cluster = i;
				firstdistance = dist;
			}
		}
		__syncthreads();

		points[tid] = sh_points[threadPosInBlock];
		
		__syncthreads();
	}
}
